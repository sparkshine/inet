#include "hip/hip_runtime.h"
//
// Copyright (C) 2013 OpenSim Ltd.
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU Lesser General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with this program; if not, see <http://www.gnu.org/licenses/>.
//

#include "CUDARadioChannel.h"
#include "ScalarImplementation.h"

#define cudaErrorCheck(code) { gpuAssert((code), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line)
{
   if (code != hipSuccess) 
      throw cRuntimeError("CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
}

// TODO: for fingerprint equality: (document these somewhere) 
// TODO:  - add -O0 compiler and -prec-sqrt=true compiler flags,
// TODO:  - use int64_t instead of double for simulation times
// TODO: extract and share parts that are common with the CPU based implementation
__global__ void computeReceptions(
        double timeScale, int transmissionCount, int radioCount,
        double propagationSpeed, double alpha,
        cuda_simtime_t *transmissionStartTimes, double *transmissionPositionXs, double *transmissionPositionYs, double *transmissionPositionZs, double *transmissionPowers, double *transmissionCarrierFrequencies,
        double *receptionPositionXs, double *receptionPositionYs, double *receptionPositionZs,
        double *receptionPowers, cuda_simtime_t *receptionStartTimes)
{
    int transmissionIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int radioIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (transmissionIndex < transmissionCount && radioIndex < radioCount)
    {
        int receptionIndex = radioIndex + radioCount * transmissionIndex;
        double dx = transmissionPositionXs[transmissionIndex] - receptionPositionXs[radioIndex];
        double dy = transmissionPositionYs[transmissionIndex] - receptionPositionYs[radioIndex];
        double dz = transmissionPositionZs[transmissionIndex] - receptionPositionZs[radioIndex];
        double distance = sqrt(dx * dx + dy * dy + dz * dz);

        double waveLength = propagationSpeed / transmissionCarrierFrequencies[transmissionIndex];
        // NOTE: this check allows to get the same result from the GPU and the CPU when the alpha is exactly 2
        double raisedDistance = alpha == 2.0 ? distance * distance : pow(distance, alpha);
        double pathLoss = distance == 0.0 ? 1.0 : waveLength * waveLength / (16.0 * M_PI * M_PI * raisedDistance);
        double receptionPower = pathLoss * transmissionPowers[transmissionIndex];
        receptionPowers[receptionIndex] = receptionPower;

        cuda_simtime_t propagationTime = distance / propagationSpeed * timeScale;
        cuda_simtime_t receptionTime = transmissionStartTimes[transmissionIndex] + propagationTime;
        receptionStartTimes[receptionIndex] = receptionTime;
    }
}

__global__ void computeSNRMinimums(
        int transmissionCount, int radioCount,
        cuda_simtime_t *transmissionDurations, double *receptionPowers, cuda_simtime_t *receptionStartTimes, double backgroundNoisePower,
        double *snrMinimums)
{
    int receptionCount = transmissionCount * radioCount;
    int candidateTransmissionIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int candidateRadioIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int candidateReceptionIndex = candidateRadioIndex + radioCount * candidateTransmissionIndex;
    cuda_simtime_t candidateTransmissionDuration = transmissionDurations[candidateTransmissionIndex];
    cuda_simtime_t candidateReceptionStartTime = receptionStartTimes[candidateReceptionIndex];
    cuda_simtime_t candidateReceptionEndTime = candidateReceptionStartTime + candidateTransmissionDuration;
    double maximumNoisePower = 0;
    for (int otherReceptionIndex = candidateRadioIndex; otherReceptionIndex < receptionCount; otherReceptionIndex += radioCount)
    {
        int otherTransmissionIndex = otherReceptionIndex / radioCount;
        cuda_simtime_t otherTransmissionDuration = transmissionDurations[otherTransmissionIndex];
        cuda_simtime_t otherReceptionStartTime = receptionStartTimes[otherReceptionIndex];
        cuda_simtime_t otherReceptionEndTime = otherReceptionStartTime + otherTransmissionDuration;
        bool isOtherStartOverlapping = candidateReceptionStartTime <= otherReceptionStartTime && otherReceptionStartTime <= candidateReceptionEndTime;
        bool isOtherEndOverlapping = candidateReceptionStartTime <= otherReceptionEndTime && otherReceptionEndTime <= candidateReceptionEndTime;
        if (isOtherStartOverlapping || isOtherEndOverlapping)
        {
            double startNoisePower = backgroundNoisePower;
            double endNoisePower = backgroundNoisePower;
            for (int noiseReceptionIndex = candidateRadioIndex; noiseReceptionIndex < receptionCount; noiseReceptionIndex += radioCount)
            {
                if (noiseReceptionIndex != candidateReceptionIndex)
                {
                    int noiseTransmissionIndex = noiseReceptionIndex / radioCount;
                    cuda_simtime_t noiseTransmissionDuration = transmissionDurations[noiseTransmissionIndex];
                    cuda_simtime_t noiseReceptionStartTime = receptionStartTimes[noiseReceptionIndex];
                    cuda_simtime_t noiseReceptionEndTime = noiseReceptionStartTime + noiseTransmissionDuration;
                    double noisePower = receptionPowers[noiseReceptionIndex];
                    if (isOtherStartOverlapping && noiseReceptionStartTime <= otherReceptionStartTime && otherReceptionStartTime <= noiseReceptionEndTime)
                        startNoisePower += noisePower;
                    if (isOtherEndOverlapping && noiseReceptionStartTime <= otherReceptionEndTime && otherReceptionEndTime <= noiseReceptionEndTime)
                        endNoisePower += noisePower;
                }
            }
            if (isOtherStartOverlapping && startNoisePower > maximumNoisePower)
                maximumNoisePower = startNoisePower;
            if (isOtherEndOverlapping && endNoisePower > maximumNoisePower)
                maximumNoisePower = endNoisePower;
        }
    }
    double candidateNoisePower = receptionPowers[candidateReceptionIndex];
    snrMinimums[candidateReceptionIndex] = candidateNoisePower / maximumNoisePower;
}

void CUDARadioChannel::computeCache(const std::vector<const XIRadio *> *radios, const std::vector<const IRadioSignalTransmission *> *transmissions)
{
    // for all transmissions compute the start and end reception times at the start and end reception positions
    int transmissionCount = transmissions->size();
    int radioCount = radios->size();
    int receptionCount = transmissionCount * radioCount;
    int transmissionSize = transmissionCount * sizeof(double);
    int radioSize = radioCount * sizeof(double);
    int receptionSize = receptionCount * sizeof(double);
    double timeScale = (double) SimTime::getScale();
    double alpha = check_and_cast<const ScalarRadioSignalFreeSpaceAttenuation *>(attenuation)->getAlpha();
    double backgroundNoisePower = check_and_cast<const ScalarRadioBackgroundNoise *>(backgroundNoise)->getPower();
    EV_DEBUG << "Radio channel is computing cache with transmission count: " << transmissionCount << ", reception count: " << radioCount << ", arrival count: " << receptionCount << endl;

    // allocate host memory
    EV_DEBUG << "Allocating host memory" << endl;
    cuda_simtime_t *hostTransmissionStartTimes = new cuda_simtime_t[transmissionCount];
    cuda_simtime_t *hostTransmissionDurations = new cuda_simtime_t[transmissionCount];
    double *hostTransmissionPositionXs = new double[transmissionCount];
    double *hostTransmissionPositionYs = new double[transmissionCount];
    double *hostTransmissionPositionZs = new double[transmissionCount];
    double *hostTransmissionPowers = new double[transmissionCount];
    double *hostTransmissionCarrierFrequencies = new double[transmissionCount];
    double *hostRadioPositionXs = new double[radioCount];
    double *hostRadioPositionYs = new double[radioCount];
    double *hostRadioPositionZs = new double[radioCount];
    double *hostReceptionPowers = new double[receptionCount];
    cuda_simtime_t *hostReceptionStartTimes = new cuda_simtime_t[receptionCount];
    double *hostSNRMinimums = new double[receptionCount];

    // prepare host data
    EV_DEBUG << "Preparing host data" << endl;
    int index = 0;
    for (std::vector<const IRadioSignalTransmission *>::const_iterator it = transmissions->begin(); it != transmissions->end(); it++)
    {
        const IRadioSignalTransmission *transmission = *it;
        Coord startPosition = transmission->getStartPosition();
        hostTransmissionStartTimes[index] = transmission->getStartTime().raw();
        hostTransmissionDurations[index] = transmission->getDuration().raw();
        hostTransmissionPositionXs[index] = startPosition.x;
        hostTransmissionPositionYs[index] = startPosition.y;
        hostTransmissionPositionZs[index] = startPosition.z;
        const ScalarRadioSignalTransmission *scalarTransmission = check_and_cast<const ScalarRadioSignalTransmission *>(transmission);
        hostTransmissionPowers[index] = scalarTransmission->getPower();
        hostTransmissionCarrierFrequencies[index] = scalarTransmission->getCarrierFrequency();
        index++;
    }
    index = 0;
    for (std::vector<const XIRadio *>::const_iterator it = radios->begin(); it != radios->end(); it++)
    {
        const XIRadio *radio = *it;
        Coord startPosition = radio->getAntenna()->getMobility()->getCurrentPosition();
        hostRadioPositionXs[index] = startPosition.x;
        hostRadioPositionYs[index] = startPosition.y;
        hostRadioPositionZs[index] = startPosition.z;
        index++;
    }

    // allocate device memory
    EV_DEBUG << "Allocating device memory" << endl;
    cuda_simtime_t *deviceTransmissionStartTimes;
    cuda_simtime_t *deviceTransmissionDurations;
    double *deviceTransmissionPositionXs;
    double *deviceTransmissionPositionYs;
    double *deviceTransmissionPositionZs;
    double *deviceTransmissionPowers;
    double *deviceTransmissionCarrierFrequencies;
    double *deviceRadioPositionXs;
    double *deviceRadioPositionYs;
    double *deviceRadioPositionZs;
    double *deviceReceptionPowers;
    cuda_simtime_t *deviceReceptionStartTimes;
    double *deviceSNRMinimums;
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionStartTimes, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionDurations, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionPositionXs, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionPositionYs, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionPositionZs, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionPowers, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceTransmissionCarrierFrequencies, transmissionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceRadioPositionXs, radioSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceRadioPositionYs, radioSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceRadioPositionZs, radioSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceReceptionPowers, receptionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceReceptionStartTimes, receptionSize)); 
    cudaErrorCheck(hipMalloc((void**)&deviceSNRMinimums, receptionSize)); 

    // copy data from host to device
    EV_DEBUG << "Copying host data to device memory" << endl;
    cudaErrorCheck(hipMemcpy(deviceTransmissionStartTimes, hostTransmissionStartTimes, transmissionSize, hipMemcpyHostToDevice)); 
    cudaErrorCheck(hipMemcpy(deviceTransmissionDurations, hostTransmissionDurations, transmissionSize, hipMemcpyHostToDevice)); 
    cudaErrorCheck(hipMemcpy(deviceTransmissionPositionXs, hostTransmissionPositionXs, transmissionSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceTransmissionPositionYs, hostTransmissionPositionYs, transmissionSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceTransmissionPositionZs, hostTransmissionPositionZs, transmissionSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceTransmissionPowers, hostTransmissionPowers, transmissionSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceTransmissionCarrierFrequencies, hostTransmissionCarrierFrequencies, transmissionSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceRadioPositionXs, hostRadioPositionXs, radioSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceRadioPositionYs, hostRadioPositionYs, radioSize, hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(deviceRadioPositionZs, hostRadioPositionZs, radioSize, hipMemcpyHostToDevice));

    // start the computation on the device
    EV_DEBUG << "Starting computation on device" << endl;
    dim3 blockSize;
    blockSize.x = 4;
    blockSize.y = 4;
    dim3 gridSize;
    gridSize.x = transmissionCount / blockSize.x + 1;
    gridSize.y = radioCount / blockSize.y + 1;

    computeReceptions<<<gridSize, blockSize>>>(
            timeScale, transmissionCount, radioCount,
            propagationSpeed, alpha, 
            deviceTransmissionStartTimes, deviceTransmissionPositionXs, deviceTransmissionPositionYs, deviceTransmissionPositionZs, deviceTransmissionPowers, deviceTransmissionCarrierFrequencies,
            deviceRadioPositionXs, deviceRadioPositionYs, deviceRadioPositionZs,
            deviceReceptionPowers, deviceReceptionStartTimes);
    
    cudaErrorCheck(hipDeviceSynchronize());

    computeSNRMinimums<<<gridSize, blockSize>>>(
            transmissionCount, radioCount,
            deviceTransmissionDurations, deviceReceptionPowers, deviceReceptionStartTimes, backgroundNoisePower,
            deviceSNRMinimums);

    // copy data from device to host
    EV_DEBUG << "Copying device data to host memory" << endl;
    cudaErrorCheck(hipMemcpy(hostReceptionPowers, deviceReceptionPowers, receptionSize, hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(hostReceptionStartTimes, deviceReceptionStartTimes, receptionSize, hipMemcpyDeviceToHost)); 
    cudaErrorCheck(hipMemcpy(hostSNRMinimums, deviceSNRMinimums, receptionSize, hipMemcpyDeviceToHost)); 

    EV_DEBUG << "Reception times: ";
    for (int receptionIndex = 0; receptionIndex < receptionCount; receptionIndex++) {
        simtime_t receptionStartTime;
        receptionStartTime.setRaw(hostReceptionStartTimes[receptionIndex]);
        EV_DEBUG << receptionStartTime << " ";
    }
    EV_DEBUG << endl;

    EV_DEBUG << "Reception powers: ";
    for (int receptionIndex = 0; receptionIndex < receptionCount; receptionIndex++)
        EV_DEBUG << hostReceptionPowers[receptionIndex] << " ";
    EV_DEBUG << endl;

    EV_DEBUG << "SNR minimums: ";
    for (int receptionIndex = 0; receptionIndex < receptionCount; receptionIndex++) {
        int radioIndex = receptionIndex % radioCount;
        int transmissionIndex = receptionIndex / radioCount;
        const XIRadio *radio = radios->at(radioIndex);
        const IRadioSignalTransmission *transmission = transmissions->at(transmissionIndex);
        simtime_t receptionStartTime;
        simtime_t receptionEndTime;
        receptionStartTime.setRaw(hostReceptionStartTimes[receptionIndex]);
        receptionEndTime.setRaw(receptionStartTime.raw() + transmission->getDuration().raw());
        double receptionPower = hostReceptionPowers[receptionIndex];
        const ScalarRadioSignalTransmission *scalarTransmission = check_and_cast<const ScalarRadioSignalTransmission *>(transmission);
        // TODO: add reception coordinates
        ScalarRadioSignalReception *reception = new ScalarRadioSignalReception(radio, transmission, receptionStartTime, receptionEndTime, Coord(), Coord(), receptionPower, scalarTransmission->getCarrierFrequency(), scalarTransmission->getBandwidth());
        double snrMinimum = hostSNRMinimums[receptionIndex];
        double snrThreshold = check_and_cast<const ScalarSNRRadioDecider *>(radio->getDecider())->getSNRThreshold();
        ScalarRadioDecision *decision = new ScalarRadioDecision(reception, snrMinimum > snrThreshold, snrMinimum);
        setCachedDecision(radio, transmission, decision);
        EV_DEBUG << snrMinimum << " ";
    }
    EV_DEBUG << endl;

    // release resources
    EV_DEBUG << "Freeing device memory" << endl;
    cudaErrorCheck(hipFree(deviceTransmissionStartTimes));
    cudaErrorCheck(hipFree(deviceTransmissionDurations));
    cudaErrorCheck(hipFree(deviceTransmissionPositionXs));
    cudaErrorCheck(hipFree(deviceTransmissionPositionYs));
    cudaErrorCheck(hipFree(deviceTransmissionPositionZs));
    cudaErrorCheck(hipFree(deviceTransmissionPowers));
    cudaErrorCheck(hipFree(deviceTransmissionCarrierFrequencies));
    cudaErrorCheck(hipFree(deviceRadioPositionXs));
    cudaErrorCheck(hipFree(deviceRadioPositionYs));
    cudaErrorCheck(hipFree(deviceRadioPositionZs));
    cudaErrorCheck(hipFree(deviceReceptionPowers));
    cudaErrorCheck(hipFree(deviceReceptionStartTimes));
    cudaErrorCheck(hipFree(deviceSNRMinimums));

    EV_DEBUG << "Freeing host memory" << endl;
    delete hostTransmissionStartTimes;
    delete hostTransmissionDurations;
    delete hostTransmissionPositionXs;
    delete hostTransmissionPositionYs;
    delete hostTransmissionPositionZs;
    delete hostTransmissionPowers;
    delete hostTransmissionCarrierFrequencies;
    delete hostRadioPositionXs;
    delete hostRadioPositionYs;
    delete hostRadioPositionZs;
    delete hostReceptionPowers;
    delete hostReceptionStartTimes;
    delete hostSNRMinimums;
}

void CUDARadioChannel::transmitSignal(const XIRadio *radio, const IRadioSignalTransmission *transmission)
{
    RadioChannel::transmitSignal(radio, transmission);
}

const IRadioDecision *CUDARadioChannel::receiveSignal(const XIRadio *radio, const IRadioSignalTransmission *transmission) const
{
    // TODO: use a background thread?
    if (cachedDecisions.size() == 0)
        const_cast<CUDARadioChannel *>(this)->computeCache((const std::vector<const XIRadio *> *)(&radios), (const std::vector<const IRadioSignalTransmission *> *)(&transmissions));
    return CachedRadioChannel::receiveSignal(radio, transmission);
}
